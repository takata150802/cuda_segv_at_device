
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cassert>
#include <string>
#include <sstream>

#define CHECK(call)                                                  \
{                                                                    \
    const hipError_t error = call;                                  \
    std::stringstream ss;                                            \
    if (error != hipSuccess)                                        \
    {                                                                \
        ss        << "CHECK cudaError_t: "                           \
                  << __FILE__                                        \
                  << "("                                             \
                  << __LINE__                                        \
                  << ")"                                             \
                  << ": "                                            \
                  << "Error"                                         \
                  << std::endl;                                      \
        ss        << "code: "                                        \
                  << error                                           \
                  << ", "                                            \
                  << "reason: "                                      \
                  << hipGetErrorString(error)                       \
                  << std::endl;                                      \
        std::cerr << ss.str();                                       \
        std::exit(EXIT_FAILURE);                                     \
    }                                                                \
}

__global__
void tmp(long n, long a, long *y)
{
    long i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a + i;
    if (i < n) y[i+n-3] = a + i;
}

int main(void)
{
    CHECK(hipGetLastError ());

    constexpr long K = 13;
    constexpr long N = 1<<5;

    std::vector< std::vector<long> > h_y (K, std::vector<long>(N, -1));
    std::vector<long*> d_y(K);
    std::vector<hipStream_t> stream(K);

    for (long i = 0; i < K; ++i) {
        size_t size_ = h_y[i].size() * sizeof(h_y[i][0]);
        CHECK(hipMalloc(&d_y[i], size_));
        CHECK(hipStreamCreate(&stream[i]));
    }

    for (long i = 0; i < K; ++i) {
        tmp<<<(N+255)/256, 256, 0, stream[i]>>>(N, i, d_y[i]);
    }

    for (long i = 0; i < K; ++i) {
        size_t size_ = h_y[i].size() * sizeof(h_y[i][0]);
        CHECK(hipMemcpy(h_y[i].data(), d_y[i], size_, hipMemcpyDeviceToHost));
    }

    for (long j = 0; j < N; ++j) {
        for (long i = 0; i < K; ++i) {
            if (h_y[i][j] == (i + j)) {
            } else {
                std::cout << "NG:";
            }
            std::cout << h_y[i][j] << ", ";
        }
        std::cout << std::endl;
    }
    CHECK(hipGetLastError ());
    return 0;
}
